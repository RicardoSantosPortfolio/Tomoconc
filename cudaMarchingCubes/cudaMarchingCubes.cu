#include <hip/hip_runtime.h>

#include <stdio.h>
#include <string.h>


//Declarando a estrutura que recebe o ponto com coordenadas e cor
struct pontoXYZcor
{
	float x, y, z;
	int RGBa;
};

__global__ void marchingCubes(int* entrada, int* saida, int width, int height, int threshold)
{
	/*Aqui, cada coluna � executada por uma thread e cada linha por um bloco, ent�o:

	bloco 0, thread 0 executa: img[0][0];
	bloco 0, thread 1 executa: img[0][1];
	bloco 0, thread 2 executa: img[0][2];

	...

	bloco n, thread 0 executa: img[n][0];
	bloco n, thread 1 executa: img[n][1];
	bloco n, thread 2 executa: img[n][2];

	...

	bloco n, thread m-2 executa: img[n][m-2];
	bloco n, thread m-1 executa: img[n][m-1];
	bloco n, thread m executa: img[n][m].

	Sendo m o n�mero de colunas e n o n�mero de linhas da imagem

	*/
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if(i < width * height)
	{
		int R = (entrada[i] & 0x00FFFFFF) >> 16;
		int G = (entrada[i] & 0x0000FFFF) >> 8;
		int B = (entrada[i] & 0x000000FF);
		int gs = ((R)*0.3)+((G)*0.59)+((B)*0.11);

		if ( gs >= threshold )
			saida[i] = entrada[i];
		else
			saida[i] = 0xFF000000;
	}

	return;
}

extern "C"
{

	__declspec(dllexport) int* cudaMarchingCubes(int* entrada, int width, int height, int threshold)
	{
		int* i;

		//Declarando as vari�veis do device
		int *d_entrada, *d_saida;
		
		i = (int*)malloc(width*height*sizeof(int));

		//Alocando as vari�veis do device
		hipMalloc((void**)&d_entrada, width*height*sizeof(int));
		hipMalloc((void**)&d_saida, width*height*sizeof(int));

		//Inicializar vari�veis CUDA
		hipMemcpy(d_entrada, entrada, width*height*sizeof(int), hipMemcpyHostToDevice);

		marchingCubes<<<width,height>>>(d_entrada, d_saida, width, height, threshold);

		//Copiar retorno das vari�veis CUDA
		hipMemcpy(i, d_saida, width*height*sizeof(int), hipMemcpyDeviceToHost);

		//Liberar as vari�veis CUDA
		hipFree(d_entrada);
		hipFree(d_saida);

		hipDeviceReset();

		//Retornar o resultado do processamento
		//memcpy(saida, i, sizeof(i));
		return i;
	}

}

